#include <cstdio>
#include <hip/hip_runtime.h>

static void displayDeviceProperties(hipDeviceProp_t *devProps);

int main()
{
    int devCount;
    hipGetDeviceCount(&devCount);

    std::printf("CUDA device count: %d\n", devCount);
    for(int i = 0; i < devCount; ++i)
    {
        hipDeviceProp_t devProps;
        hipGetDeviceProperties(&devProps, i);
        std::printf("CUDA device properties for device %d:\n", i);
        displayDeviceProperties(&devProps);
    }
    return 0;
}

static void displayDeviceProperties(hipDeviceProp_t *devProps)
{
    std::printf("name: %s\n", devProps->name);
}
