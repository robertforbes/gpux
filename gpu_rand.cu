#include <cstdio>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

__global__ void gpurand(hiprandState* state, uint64_t seed, float* rndVals, int n)
{
    int tid = 0;
    hiprand_init(seed, tid, 0, state);
    for(int i = 0; i < n; ++i)
    {
        std::printf("hiprand_uniform %10.5f\n", hiprand_uniform(state));
    }
}

int main()
{
    hiprandState* state;
    float* devRndVals;
    // float* hostRndVals;
    int nThreads = 1;
    int N = 10;

    hipMalloc(&state, nThreads * sizeof(hiprandState));
    hipMalloc(&devRndVals, N * sizeof(float));

    gpurand<<<1,1>>>(state, 123, devRndVals, N);

    hipDeviceSynchronize();
    hipFree(state);
    hipFree(devRndVals);
    return 0;
}
