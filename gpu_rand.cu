#include <cstdio>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

__global__ void gpuRand(hiprandState* state, uint64_t seed, float* rndVals, int n)
{
    int tid = 0;
    hiprand_init(seed, tid, 0, state);
    for(int i = 0; i < n; ++i)
    {
        std::printf("hiprand_uniform %10.5f\n", hiprand_uniform(state));
    }
}

__global__ void gpuRandInt(
    hiprandState* state,
    uint64_t seed,
    int* rndVals,
    int n,
    int min,
    int max)
{
    int tid = 0;
    hiprand_init(seed, tid, 0, state);
    for(int i = 0; i < n; ++i)
    {
        float rndF = hiprand_uniform(state);
        std::printf("hiprand_uniform %10.5f\n", rndF);
        float scaled = (float)min + rndF * (float)(max - min);
        int rndI = (int)round(scaled); 
        std::printf("rand int %d\n", rndI);
    }
}

int main()
{
    hiprandState* state;
    float* devRndVals;
    int* devIntVals;
    // float* hostRndVals;
    int nThreads = 1;
    int N = 10;

    hipMalloc(&state, nThreads * sizeof(hiprandState));
    hipMalloc(&devRndVals, N * sizeof(float));
    hipMalloc(&devIntVals, N * sizeof(int));

    gpuRand<<<1,1>>>(state, 123, devRndVals, N);
    gpuRandInt<<<1,1>>>(state, 123, devIntVals, N, 3, 7);

    hipDeviceSynchronize();
    hipFree(state);
    hipFree(devRndVals);
    hipFree(devIntVals);

    return 0;
}
